#include "hip/hip_runtime.h"
#include "common.h"

#define BLOCK_SIZE 16

template <typename scalar_t>
__global__ void matmul_cuda_kernel(
    index_t m, index_t n, index_t k, const scalar_t *left,
    const scalar_t *right, scalar_t *output
) {
    // index
    index_t ty = threadIdx.y;
    index_t tx = threadIdx.x;
    index_t gy = blockIdx.y * blockDim.y + ty;
    index_t gx = blockIdx.x * blockDim.x + tx;

    // window
    scalar_t reduced = 0.0;
    for (index_t offset = 0; offset < k; offset += BLOCK_SIZE) {
        // cache
        __shared__ scalar_t cache_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ scalar_t cache_b[BLOCK_SIZE][BLOCK_SIZE];

        // store
        cache_a[ty][tx] = left[gy * k + (offset + tx)];
        cache_b[ty][tx] = right[(offset + ty) * n + gx];
        __syncthreads();

        // product
        for (index_t i = 0; i < BLOCK_SIZE; i += 1) {
            reduced += cache_a[ty][i] * cache_b[i][tx];
        }
        __syncthreads();
    }

    // store
    output[gy * n + gx] = reduced;
}

torch::Tensor matmul_cuda(
    const torch::Tensor &left, const torch::Tensor &right
) {
    CHECK_DIM(left, 2);
    CHECK_DIM(right, 2);
    TORCH_CHECK(left.size(1) == right.size(0));
    TORCH_CHECK(left.scalar_type() == right.scalar_type());

    // sizes
    index_t m = left.size(0);
    index_t n = right.size(1);
    index_t k = left.size(1);
    TORCH_CHECK(m % BLOCK_SIZE == 0);
    TORCH_CHECK(n % BLOCK_SIZE == 0);
    TORCH_CHECK(k % BLOCK_SIZE == 0);
    auto output = torch::zeros({m, n}, left.options());

    // dispatch
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(n / BLOCK_SIZE, m / BLOCK_SIZE);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        left.scalar_type(), "matmul_cuda_kernel", ([&] {
            matmul_cuda_kernel<scalar_t><<<blocks, threads>>>(
                m, n, k, left.data_ptr<scalar_t>(), right.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>()
            );
            TORCH_CHECK(hipGetLastError() == hipSuccess);
        })
    );

    //
    return output;
}