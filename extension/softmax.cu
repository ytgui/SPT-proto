#include "hip/hip_runtime.h"
#include "common.h"

#define TSZ 4
#define BSZ 16

// clang-format off
template <typename scalar_t, typename vector_t>
__global__ void softmax_forward_kernel(
    index_t seq_length, index_t nonzeros, const index_t *indptr,
    const index_t *indices, const scalar_t *values, scalar_t *output) {
    // index
    index_t ty = threadIdx.y;
    index_t gy = blockIdx.y * blockDim.y + ty;
    index_t gz = blockIdx.z * blockDim.z;

    // cumulate
    scalar_t cumulated = 0.0;
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += TSZ) {
        vector_t cache = __ldg(
            (const vector_t *)&values[gz * nonzeros + i]
        );
        index_vt index = __ldg(
            (const index_vt *)&indices[gz * nonzeros + i]
        );
        cumulated += expf(cache.x) * (index.x <= gy);
        cumulated += expf(cache.y) * (index.y <= gy);
        cumulated += expf(cache.z) * (index.z <= gy);
        cumulated += expf(cache.w) * (index.w <= gy);
    }
    cumulated = fmax(1e-9, cumulated);

    // softmax
    scalar_t scale = 1.0 / cumulated;
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += TSZ) {
        vector_t cache = __ldg(
            (const vector_t *)&values[gz * nonzeros + i]
        );
        index_vt index = __ldg(
            (const index_vt *)&indices[gz * nonzeros + i]
        );
        cache.x = scale * expf(cache.x) * (index.x <= gy);
        cache.y = scale * expf(cache.y) * (index.y <= gy);
        cache.z = scale * expf(cache.z) * (index.z <= gy);
        cache.w = scale * expf(cache.w) * (index.w <= gy);
        __stcs((vector_t *)&output[gz * nonzeros + i], cache);
    }
}

template <typename scalar_t, typename vector_t>
__global__ void softmax_backward_kernel(
    index_t seq_length, index_t nonzeros, const index_t *indptr,
    const index_t *indices, const scalar_t *output,
    const scalar_t *grad_output, scalar_t *grad_values) {
    // index
    index_t ty = threadIdx.y;
    index_t gy = blockIdx.y * blockDim.y + ty;
    index_t gz = blockIdx.z * blockDim.z;

    // cumulate
    scalar_t cumulated = 0.0;
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += TSZ) {
        vector_t cache_output = __ldg((const vector_t *)&output[gz * nonzeros + i]);
        vector_t cache_grad_output = __ldg((const vector_t *)&grad_output[gz * nonzeros + i]);
        cumulated += cache_output.x * cache_grad_output.x * (indices[gz * nonzeros + i + 0] <= gy);
        cumulated += cache_output.y * cache_grad_output.y * (indices[gz * nonzeros + i + 1] <= gy);
        cumulated += cache_output.z * cache_grad_output.z * (indices[gz * nonzeros + i + 2] <= gy);
        cumulated += cache_output.w * cache_grad_output.w * (indices[gz * nonzeros + i + 3] <= gy);
    }
    cumulated = fmax(1e-9, cumulated);

    // gradient
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += TSZ) {
        vector_t cache_output = __ldg((const vector_t *)&output[gz * nonzeros + i]);
        vector_t cache_grad_output = __ldg((const vector_t *)&grad_output[gz * nonzeros + i]);
        cache_output.x = cache_output.x * (cache_grad_output.x - cumulated) * (indices[gz * nonzeros + i + 0] <= gy);
        cache_output.y = cache_output.y * (cache_grad_output.y - cumulated) * (indices[gz * nonzeros + i + 1] <= gy);
        cache_output.z = cache_output.z * (cache_grad_output.z - cumulated) * (indices[gz * nonzeros + i + 2] <= gy);
        cache_output.w = cache_output.w * (cache_grad_output.w - cumulated) * (indices[gz * nonzeros + i + 3] <= gy);
        __stcs((vector_t *)&grad_values[gz * nonzeros + i], cache_output);
    }
}
// clang-format on

torch::Tensor softmax_forward_cuda(
    const torch::Tensor &indptr, const torch::Tensor &indices,
    const torch::Tensor &values
) {
    CHECK_DIM(indptr, 1);
    CHECK_DIM(indices, 2);
    CHECK_DIM(values, 2);
    CHECK_TYPE(indptr, torch::kInt32);
    CHECK_TYPE(indices, torch::kInt32);
    TORCH_CHECK(indices.sizes() == values.sizes());

    // sizes
    index_t nonzeros = indices.size(-1);
    index_t batch_size = indices.size(0);
    index_t seq_length = indptr.size(-1) - 1;
    TORCH_CHECK((seq_length % BSZ) == 0);
    auto output = torch::zeros_like(values);

    // dispatch
    dim3 threads(1, BSZ);
    dim3 blocks(1, seq_length / BSZ, batch_size);
    softmax_forward_kernel<float, float4><<<blocks, threads>>>(
        seq_length, nonzeros, indptr.data_ptr<index_t>(),
        indices.data_ptr<index_t>(), values.data_ptr<float>(),
        output.data_ptr<float>()
    );
    TORCH_CHECK(hipGetLastError() == hipSuccess);

    //
    return output;
}

torch::Tensor softmax_backward_cuda(
    const torch::Tensor &indptr, const torch::Tensor &indices,
    const torch::Tensor &output, const torch::Tensor &grad_output
) {
    CHECK_DIM(indptr, 1);
    CHECK_DIM(indices, 2);
    CHECK_DIM(output, 2);
    CHECK_DIM(grad_output, 2);
    CHECK_TYPE(indptr, torch::kInt32);
    CHECK_TYPE(indices, torch::kInt32);
    TORCH_CHECK(grad_output.sizes() == output.sizes());
    TORCH_CHECK(indices.sizes() == output.sizes());

    // sizes
    index_t nonzeros = indices.size(-1);
    index_t batch_size = indices.size(0);
    index_t seq_length = indptr.size(-1) - 1;
    TORCH_CHECK((seq_length % BSZ) == 0);
    auto grad_values = torch::zeros_like(output);

    // dispatch
    dim3 threads(1, BSZ);
    dim3 blocks(1, seq_length / BSZ, batch_size);
    softmax_backward_kernel<float, float4><<<blocks, threads>>>(
        seq_length, nonzeros, indptr.data_ptr<index_t>(),
        indices.data_ptr<index_t>(), output.data_ptr<float>(),
        grad_output.data_ptr<float>(), grad_values.data_ptr<float>()
    );
    TORCH_CHECK(hipGetLastError() == hipSuccess);

    //
    return grad_values;
}
