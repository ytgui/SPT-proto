#include "hip/hip_runtime.h"
#include "common.h"

#define BK 16
#define BM 16

template <typename scalar_t>
__global__ void sddmm_forward_cuda_kernel(
    index_t seq_length, index_t d_head, const index_t *indptr,
    const index_t *indices, const scalar_t *query, const scalar_t *key,
    scalar_t *output
) {
    // index
    index_t local_y = threadIdx.x;
    index_t global_y = blockIdx.x * blockDim.x;
    index_t row = global_y + local_y;

    // contract
    for (index_t i = indptr[row]; i < indptr[row + 1]; i += 1) {
        index_t col = indices[i];

        // product
        scalar_t reduced = 0.0;
        for (index_t k = 0; k < d_head; k += 1) {
            reduced += query[row * d_head + k] * key[col * d_head + k];
        }

        // store
        output[i] = reduced;
    }
}

torch::Tensor sddmm_forward_cuda(
    const torch::Tensor &indptr, const torch::Tensor &indices,
    const torch::Tensor &query, const torch::Tensor &key
) {
    CHECK_DIM(key, 2);
    CHECK_DIM(query, 2);
    CHECK_DIM(indptr, 1);
    CHECK_DIM(indices, 1);
    CHECK_TYPE(indptr, torch::kInt64);
    CHECK_TYPE(indices, torch::kInt64);
    TORCH_CHECK(query.sizes() == key.sizes());
    TORCH_CHECK(query.scalar_type() == key.scalar_type());

    // sizes
    index_t d_head = query.size(-1);
    TORCH_CHECK((d_head % BK) == 0);
    index_t seq_length = query.size(0);
    TORCH_CHECK((seq_length % BM) == 0);
    TORCH_CHECK(indptr.size(0) == seq_length + 1);
    auto output = torch::zeros_like(indices, query.options());

    // dispatch
    dim3 threads(BM);
    dim3 blocks(seq_length / BM);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        query.scalar_type(), "sddmm_forward_cuda_kernel", ([&] {
            sddmm_forward_cuda_kernel<scalar_t><<<blocks, threads>>>(
                seq_length, d_head, indptr.data_ptr<index_t>(),
                indices.data_ptr<index_t>(), query.data_ptr<scalar_t>(),
                key.data_ptr<scalar_t>(), output.data_ptr<scalar_t>()
            );
            TORCH_CHECK(hipGetLastError() == hipSuccess);
        })
    );

    //
    return output;
}