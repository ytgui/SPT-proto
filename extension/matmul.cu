#include "hip/hip_runtime.h"
#include "common.h"

#define TM 8
#define TN 8
#define BK 16
#define BM 128
#define BN 128

// clang-format off
template <typename scalar_t>
__global__ void matmul_cuda_kernel(
    index_t m, index_t n, index_t k, const scalar_t *left,
    const scalar_t *right, scalar_t *output
) {
    // index
    index_t thread_y = threadIdx.y;
    index_t thread_x = threadIdx.x;
    index_t global_y = blockIdx.y * BM;
    index_t global_x = blockIdx.x * BN;

    // cache
    __shared__ scalar_t cache_a[BM][BK];
    __shared__ scalar_t cache_b[BK][BN];

    // window
    scalar_t local_a[TM] = {0.0};
    scalar_t local_b[TN] = {0.0};
    scalar_t reduced[TM][TN] = {0.0};
    for (index_t offset_k = 0; offset_k < k; offset_k += BK) {
        // load
        for (index_t tile_y = 0; tile_y < TM; tile_y += 1) {
            index_t local_y = tile_y * blockDim.y + thread_y;
            cache_a[local_y][thread_x] = left[
                (global_y + local_y) * k + (offset_k + thread_x)
            ];
        }
        for (index_t tile_x = 0; tile_x < TN; tile_x += 1) {
            index_t local_x = tile_x * blockDim.x + thread_x;
            cache_b[thread_y][local_x] = right[
                (offset_k + thread_y) * n + (global_x + local_x)
            ];
        }
        __syncthreads();

        // reduce
        for (index_t i = 0; i < BK; i += 1) {
            // tile a
            for (index_t tile_y = 0; tile_y < TM; tile_y += 1) {
                index_t local_y = tile_y * blockDim.y + thread_y;
                local_a[tile_y] = cache_a[local_y][i];
            }
            // tile b
            for (index_t tile_x = 0; tile_x < TN; tile_x += 1) {
                index_t local_x = tile_x * blockDim.x + thread_x;
                local_b[tile_x] = cache_b[i][local_x];
            }
            // product
            for (index_t tile_y = 0; tile_y < TM; tile_y += 1) {
                for (index_t tile_x = 0; tile_x < TN; tile_x += 1) {
                    reduced[tile_y][tile_x] += local_a[tile_y] * local_b[tile_x];
                }
            }
        }
        __syncthreads();
    }

    // store
    for (index_t tile_y = 0; tile_y < TM; tile_y += 1) {
        for (index_t tile_x = 0; tile_x < TN; tile_x += 1) {
            index_t local_y = tile_y * blockDim.y + thread_y;
            index_t local_x = tile_x * blockDim.x + thread_x;
            output[
                (global_y + local_y) * n + (global_x + local_x)
            ] = reduced[tile_y][tile_x];
        }
    }
}
// clang-format on

torch::Tensor matmul_cuda(
    const torch::Tensor &left, const torch::Tensor &right
) {
    CHECK_DIM(left, 2);
    CHECK_DIM(right, 2);
    TORCH_CHECK(left.size(1) == right.size(0));
    TORCH_CHECK(left.scalar_type() == right.scalar_type());

    // sizes
    index_t m = left.size(0);
    index_t n = right.size(1);
    index_t k = left.size(1);
    TORCH_CHECK(k % BK == 0);
    TORCH_CHECK(n % BN == 0);
    TORCH_CHECK(m % BM == 0);
    auto output = torch::zeros({m, n}, left.options());

    // dispatch
    dim3 blocks(n / BN, m / BM);
    dim3 threads(BN / TN, BM / TM);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        left.scalar_type(), "matmul_cuda_kernel", ([&] {
            matmul_cuda_kernel<scalar_t><<<blocks, threads>>>(
                m, n, k, left.data_ptr<scalar_t>(), right.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>()
            );
            TORCH_CHECK(hipGetLastError() == hipSuccess);
        })
    );

    //
    return output;
}
