#include "hip/hip_runtime.h"
#include "common.h"

#define BLOCK_SIZE 16
#define WORKER_SIZE 4

using vector_t = int4;

// clang-format off
template <unsigned N_SPACES, unsigned N_COLS>
__global__ void lookup_forward_kernel(
    index_t batch_size, index_t seq_length, index_t nonzeros,
    const index_t *indptr, const index_t *left, const index_t *right,
    index_t *output) {
    // index
    index_t ty = threadIdx.y;
    index_t tx = threadIdx.x;
    index_t gz = blockIdx.z * blockDim.z;
    index_t gy = blockIdx.y * blockDim.y + ty;

    // load lhs
    __shared__ uint16_t cache_lhs[BLOCK_SIZE][N_SPACES];
    for (index_t k = tx; k < N_SPACES; k += WORKER_SIZE) {
        cache_lhs[ty][k] = left[
            gz * seq_length * N_SPACES + gy * N_SPACES + k
        ];
    }

    // output
    #define N_SLOTS 4
    index_t cursors[N_SLOTS] = {tx, tx, tx, tx};
    __shared__ uint16_t indices[BLOCK_SIZE][N_SLOTS][N_COLS];

    // window
    for (index_t offset_x = 0; offset_x < seq_length; offset_x += BLOCK_SIZE) {
        // load rhs
        __shared__ uint16_t cache_rhs[BLOCK_SIZE][N_SPACES];
        for (index_t k = tx; k < N_SPACES; k += WORKER_SIZE) {
            cache_rhs[ty][k] = right[
                gz * seq_length * N_SPACES + (offset_x + ty) * N_SPACES + k
            ];
        }
        __syncthreads();

        // lookup
        for (index_t local_x = tx; local_x < BLOCK_SIZE; local_x += WORKER_SIZE) {
            // tril
            if ((offset_x + local_x) > gy) {
                break;
            }
            // count
            index_t count = 0;
            for (index_t k = 0; k < N_SPACES; k += 1) {
                count += (
                    cache_lhs[ty][k] == cache_rhs[local_x][k]
                );
            }
            index_t slot = min(
                N_SLOTS - 1, count / (N_SPACES / N_SLOTS)
            );
            index_t cursor = cursors[slot];
            indices[ty][slot][cursor] = offset_x + local_x;
            cursors[slot] = min(cursor + WORKER_SIZE, N_COLS - tx - 1);
        }
        __syncthreads();
    }

    // store
    index_t offset = indptr[gy];
    index_t slot = N_SLOTS - 1, cursor = tx;
    for (index_t local_x = tx; local_x < min(gy + 1, N_COLS); local_x += WORKER_SIZE) {
        while (cursor >= cursors[slot]) {
            slot = slot - 1; cursor = tx;
        }
        output[gz * nonzeros + offset + local_x] = indices[ty][slot][cursor];
        cursor = cursor + WORKER_SIZE;
    }
}
// clang-format on

torch::Tensor lookup_forward_cuda(
    const torch::Tensor &config, const torch::Tensor &indptr,
    const torch::Tensor &query, const torch::Tensor &key
) {
    CHECK_DIM(key, 3);
    CHECK_DIM(query, 3);
    CHECK_DIM(indptr, 1);
    CHECK_TYPE(key, torch::kInt32);
    CHECK_TYPE(query, torch::kInt32);
    CHECK_TYPE(indptr, torch::kInt32);
    TORCH_CHECK(query.sizes() == key.sizes());

    // sizes
    index_t batch_size = query.size(0);
    index_t seq_length = query.size(1);
    index_t n_subspaces = query.size(-1);
    index_t sparse_coeff = config.size(0);
    TORCH_CHECK(seq_length % BLOCK_SIZE == 0);
    TORCH_CHECK(seq_length % sparse_coeff == 0);
    index_t colsize = seq_length / sparse_coeff;
    TORCH_CHECK(colsize % BLOCK_SIZE == 0);
    index_t nonzeros = (1 + colsize) * colsize / 2 +
                       (seq_length - colsize) * colsize;
    auto output = torch::empty({batch_size, nonzeros}, query.options());

    // dispatch
    dim3 blocks(1, seq_length / BLOCK_SIZE, batch_size);
    if (n_subspaces == 8 && colsize == 64) {
        dim3 threads(WORKER_SIZE, BLOCK_SIZE);
        lookup_forward_kernel<8, 64><<<blocks, threads>>>(
            batch_size, seq_length, nonzeros, indptr.data_ptr<index_t>(),
            query.data_ptr<index_t>(), key.data_ptr<index_t>(),
            output.data_ptr<index_t>()
        );
    } else if (n_subspaces == 8 && colsize == 128) {
        dim3 threads(WORKER_SIZE, BLOCK_SIZE);
        lookup_forward_kernel<8, 128><<<blocks, threads>>>(
            batch_size, seq_length, nonzeros, indptr.data_ptr<index_t>(),
            query.data_ptr<index_t>(), key.data_ptr<index_t>(),
            output.data_ptr<index_t>()
        );
    } else {
        TORCH_CHECK(false && "n_subspaces not supported");
    }
    TORCH_CHECK(hipGetLastError() == hipSuccess);

    //
    return output;
}