#include "hip/hip_runtime.h"
#include "common.h"

#define TSZ 4
#define BSZ 16

using vector_t = int4;

// clang-format off
template <unsigned NSP, unsigned MAXNZ>
__global__ void lookup_forward_kernel(
    index_t batch_size, index_t seq_length, index_t nonzeros,
    const index_t *query, const index_t *store, index_t *output
) {
    // index
    index_t ty = threadIdx.y;
    index_t gz = blockIdx.z * blockDim.z;
    index_t gy = blockIdx.y * blockDim.y + ty;

    // cache
    index_t cache_query[NSP];
    for (int16_t k = 0; k < NSP; k += 1) {
        cache_query[k] = query[
            gz * seq_length * NSP + gy * NSP + k
        ];
    }

    // result
    #define NSLOTS 3
    int16_t cache_sizes[NSLOTS] = {};
    int16_t cache_indices[NSLOTS][MAXNZ];

    // window
    for (int16_t offset_x = 0; offset_x < seq_length; offset_x += BSZ) {
        // cache
        __shared__ index_t cache_store[BSZ][NSP];
        for (int16_t k = 0; k < NSP; k += 1) {
            cache_store[ty][k] = store[
                gz * seq_length * NSP + (offset_x + ty) * NSP + k
            ];
        }
        __syncthreads();

        // product
        for (int16_t tx = 0; tx < BSZ; tx += 1) {
            int16_t count = 0;
            for (int16_t k = 0; k < NSP; k += 1) {
                count += (cache_query[k] == cache_store[tx][k]);   
            }
            count = min(7, count) / 2;
            int16_t slot = 32 - __clz(count);
            int16_t cursor = min(cache_sizes[slot], MAXNZ - 1);
            cache_indices[slot][cursor] = offset_x + tx;
            cache_sizes[slot] += 1;
        }
        __syncthreads();
    }

    // store
    int16_t slot = NSLOTS - 1, cursor = 0;
    for (int16_t gx = 0; gx < nonzeros; gx += TSZ) {
        index_t buffer[TSZ];
        for (index_t t = 0; t < TSZ; t += 1) {
            buffer[t] = cache_indices[slot][cursor];
            bool cond = (cursor == cache_sizes[slot] - 1);
            cursor = cond ? 0 : (cursor + 1);
            slot = cond ? (slot - 1) : slot;
        }
        __stcs(
            (vector_t *)&output[
                gz * seq_length * nonzeros + gy * nonzeros + gx
            ], *(const vector_t *)buffer
        );
    }
}
// clang-format on

torch::Tensor lookup_forward_cuda(
    const torch::Tensor &config, const torch::Tensor &query,
    const torch::Tensor &store
) {
    CHECK_DIM(query, 3);
    CHECK_DIM(store, 3);
    CHECK_TYPE(query, torch::kInt32);
    CHECK_TYPE(store, torch::kInt32);
    TORCH_CHECK(query.sizes() == store.sizes());
    TORCH_CHECK(query.scalar_type() == store.scalar_type());

    // sizes
    index_t sparsity = config.size(0);
    index_t batch_size = query.size(0);
    index_t seq_length = query.size(1);
    index_t n_subspaces = query.size(-1);
    TORCH_CHECK(seq_length % BSZ == 0);
    TORCH_CHECK(n_subspaces % TSZ == 0);
    TORCH_CHECK(seq_length % sparsity == 0);
    index_t nonzeros = seq_length / sparsity;
    TORCH_CHECK(nonzeros % BSZ == 0);
    auto output = torch::empty(
        {batch_size, seq_length, nonzeros}, query.options()
    );

    // dispatch
    dim3 threads(1, BSZ);
    dim3 blocks(1, seq_length / BSZ, batch_size);
    if (n_subspaces == 8) {
        if (nonzeros <= 64) {
            lookup_forward_kernel<8, 64><<<blocks, threads>>>(
                batch_size, seq_length, nonzeros, query.data_ptr<index_t>(),
                store.data_ptr<index_t>(), output.data_ptr<index_t>()
            );
        } else if (nonzeros <= 128) {
            lookup_forward_kernel<8, 128><<<blocks, threads>>>(
                batch_size, seq_length, nonzeros, query.data_ptr<index_t>(),
                store.data_ptr<index_t>(), output.data_ptr<index_t>()
            );
        } else if (nonzeros <= 256) {
            lookup_forward_kernel<8, 256><<<blocks, threads>>>(
                batch_size, seq_length, nonzeros, query.data_ptr<index_t>(),
                store.data_ptr<index_t>(), output.data_ptr<index_t>()
            );
        } else {
            TORCH_CHECK(false && "seq_length / sparsity not supported");
        }
    } else {
        TORCH_CHECK(false && "n_subspaces not supported");
    }
    TORCH_CHECK(hipGetLastError() == hipSuccess);

    //
    return output;
}