#include "hip/hip_runtime.h"
#include "common.h"

#define BLOCK_SIZE 4

// clang-format off
template <typename scalar_t>
__global__ void cdist_forward_kernel(
    index_t n_queries, index_t n_codewords, index_t d_code,
    const scalar_t *query, const scalar_t *table, scalar_t *output) {
    // index
    index_t ty = threadIdx.y;
    index_t tx = threadIdx.x;
    index_t gz = blockIdx.z * blockDim.z;
    index_t gy = blockIdx.y * blockDim.y + ty;
    index_t gx = blockIdx.x * blockDim.x + tx;

    // window
    scalar_t reduced = 0.0;
    for (index_t offset = 0; offset < d_code; offset += BLOCK_SIZE) {
        // cache
        __shared__ scalar_t cache_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ scalar_t cache_b[BLOCK_SIZE][BLOCK_SIZE];

        // store
        cache_a[ty][tx] = query[
            gz * n_queries * d_code + gy * d_code + (offset + tx)
        ];
        cache_b[ty][tx] = table[
            gz * n_codewords * d_code + gx * d_code + (offset + ty)
        ];
        __syncthreads();

        // product
        for (index_t i = 0; i < BLOCK_SIZE; i += 1) {
            reduced += fabsf(cache_a[ty][i] - cache_b[i][tx]);
        }
        __syncthreads();
    }

    // store
    index_t offset = gz * n_queries * n_codewords;
    output[offset + gy * n_codewords + gx] = reduced;
}

template <typename scalar_t>
__global__ void cdist_backward_kernel(
    index_t n_queries, index_t n_codewords, index_t d_code,
    const scalar_t *query, const scalar_t *table, const scalar_t *grad_output,
    scalar_t *grad_query, scalar_t *grad_table) {
    // index
    index_t ty = threadIdx.y;
    index_t tx = threadIdx.x;
    index_t gz = blockIdx.z * blockDim.z;
    index_t gy = blockIdx.y * blockDim.y + ty;
    index_t gx = blockIdx.x * blockDim.x + tx;

    // load
    scalar_t value = grad_output[
        gz * n_queries * n_codewords + gy * n_codewords + gx
    ];

    // window
    for (index_t offset = 0; offset < d_code; offset += BLOCK_SIZE) {
        // cache
        __shared__ scalar_t cache_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ scalar_t cache_b[BLOCK_SIZE][BLOCK_SIZE];

        // store
        cache_a[ty][tx] = query[
            gz * n_queries * d_code + gy * d_code + (offset + tx)
        ];
        cache_b[ty][tx] = table[
            gz * n_codewords * d_code + gx * d_code + (offset + ty)
        ];
        __syncthreads();

        // product
        for (index_t i = 0; i < BLOCK_SIZE; i += 1) {
            scalar_t v = (
                cache_a[ty][i] - cache_b[i][tx]
            ) > 0 ? value : -value;
            atomicAdd(&grad_query[
                gz * n_queries * d_code + gy * d_code + (offset + i)
            ], v);
            atomicAdd(&grad_table[
                gz * n_codewords * d_code + gx * d_code + (offset + i)
            ], -v);
        }
        __syncthreads();
    }
}
// clang-format on

torch::Tensor cdist_forward_cuda(
    const torch::Tensor &query, const torch::Tensor &table
) {
    CHECK_DIM(query, 3);
    CHECK_DIM(table, 3);
    TORCH_CHECK(query.size(0) == table.size(0));
    TORCH_CHECK(query.size(-1) == table.size(-1));
    TORCH_CHECK(query.scalar_type() == table.scalar_type());

    // sizes
    index_t d_code = table.size(-1);
    index_t n_queries = query.size(1);
    index_t n_codewords = table.size(1);
    index_t n_subspaces = table.size(0);
    TORCH_CHECK(d_code % BLOCK_SIZE == 0);
    TORCH_CHECK(n_queries % BLOCK_SIZE == 0);
    TORCH_CHECK(n_codewords % BLOCK_SIZE == 0);
    auto output = torch::zeros(
        {n_subspaces, n_queries, n_codewords}, query.options()
    );

    // dispatch
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(n_codewords / BLOCK_SIZE, n_queries / BLOCK_SIZE, n_subspaces);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        query.scalar_type(), "cdist_forward_kernel", ([&] {
            cdist_forward_kernel<scalar_t><<<blocks, threads>>>(
                n_queries, n_codewords, d_code, query.data_ptr<scalar_t>(),
                table.data_ptr<scalar_t>(), output.data_ptr<scalar_t>()
            );
            TORCH_CHECK(hipGetLastError() == hipSuccess);
        })
    );

    //
    return output;
}

std::vector<torch::Tensor> cdist_backward_cuda(
    const torch::Tensor &query, const torch::Tensor &table,
    const torch::Tensor &grad_output
) {
    CHECK_DIM(query, 3);
    CHECK_DIM(table, 3);
    CHECK_DIM(grad_output, 3);
    TORCH_CHECK(query.size(0) == table.size(0));
    TORCH_CHECK(query.size(-1) == table.size(-1));
    TORCH_CHECK(query.size(0) == grad_output.size(0));
    TORCH_CHECK(query.size(1) == grad_output.size(1));
    TORCH_CHECK(table.size(1) == grad_output.size(-1));
    TORCH_CHECK(query.scalar_type() == table.scalar_type());
    TORCH_CHECK(query.scalar_type() == grad_output.scalar_type());

    // sizes
    index_t d_code = table.size(-1);
    index_t n_queries = query.size(1);
    index_t n_codewords = table.size(1);
    index_t n_subspaces = table.size(0);
    TORCH_CHECK(d_code % BLOCK_SIZE == 0);
    TORCH_CHECK(n_queries % BLOCK_SIZE == 0);
    TORCH_CHECK(n_codewords % BLOCK_SIZE == 0);
    auto grad_query = torch::zeros_like(query);
    auto grad_table = torch::zeros_like(table);

    // dispatch
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(n_codewords / BLOCK_SIZE, n_queries / BLOCK_SIZE, n_subspaces);
    AT_DISPATCH_FLOATING_TYPES(
        query.scalar_type(), "cdist_backward_kernel", ([&] {
            cdist_backward_kernel<scalar_t><<<blocks, threads>>>(
                n_queries, n_codewords, d_code, query.data_ptr<scalar_t>(),
                table.data_ptr<scalar_t>(), grad_output.data_ptr<scalar_t>(),
                grad_query.data_ptr<scalar_t>(), grad_table.data_ptr<scalar_t>()
            );
            TORCH_CHECK(hipGetLastError() == hipSuccess);
        })
    );

    //
    return {grad_query, grad_table};
}