#include "hip/hip_runtime.h"
#include "common.h"

#define BLOCK_SIZE 4

#define TSZ 4
#define BSZ 64

// clang-format off
template <typename scalar_t, typename vector_t>
__global__ void cdist_forward_kernel(
    index_t n_queries, index_t n_codewords, index_t d_code,
    const scalar_t *query, const scalar_t *table, scalar_t *output) {
    // index
    index_t ty = threadIdx.y;
    index_t gz = blockIdx.z * blockDim.z;
    index_t gy = blockIdx.y * blockDim.y + ty;

    // window
    for (index_t offset_x = 0; offset_x < n_codewords; offset_x += BSZ) {
        scalar_t reduced[BSZ] = {};
        for (index_t offset_k = 0; offset_k < d_code; offset_k += TSZ) {
            // cache
            __shared__ vector_t cache_a[BSZ];
            __shared__ vector_t cache_b[BSZ];
            cache_a[ty] = __ldg(
                (const vector_t *)&query[
                    gz * n_queries * d_code + gy * d_code + offset_k
                ]
            );
            cache_b[ty] = __ldg(
                (const vector_t *)&table[
                    gz * n_codewords * d_code + (offset_x + ty) * d_code + offset_k
                ]
            );
            __syncthreads();

            // product
            for (index_t tx = 0; tx < BSZ; tx += 1) {
                reduced[tx] += fabsf(cache_a[ty].x - cache_b[tx].x);
                reduced[tx] += fabsf(cache_a[ty].y - cache_b[tx].y);
                reduced[tx] += fabsf(cache_a[ty].z - cache_b[tx].z);
                reduced[tx] += fabsf(cache_a[ty].w - cache_b[tx].w);
            }
            __syncthreads();
        }

        // store
        index_t offset_z = gz * n_queries * n_codewords;
        for (index_t tx = 0; tx < BSZ; tx += TSZ) {
            index_t gx = offset_x + tx;
            __stcs(
                (vector_t *)&output[offset_z + gy * n_codewords + gx],
                *(const vector_t *)&reduced[tx]
            );
        }
    }
}

template <typename scalar_t>
__global__ void cdist_backward_kernel(
    index_t n_queries, index_t n_codewords, index_t d_code,
    const scalar_t *query, const scalar_t *table, const scalar_t *grad_output,
    scalar_t *grad_query, scalar_t *grad_table) {
    // index
    index_t ty = threadIdx.y;
    index_t tx = threadIdx.x;
    index_t gz = blockIdx.z * blockDim.z;
    index_t gy = blockIdx.y * blockDim.y + ty;
    index_t gx = blockIdx.x * blockDim.x + tx;

    // load
    scalar_t value = grad_output[
        gz * n_queries * n_codewords + gy * n_codewords + gx
    ];

    // window
    for (index_t offset = 0; offset < d_code; offset += BLOCK_SIZE) {
        // cache
        __shared__ scalar_t cache_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ scalar_t cache_b[BLOCK_SIZE][BLOCK_SIZE];

        // store
        cache_a[ty][tx] = query[
            gz * n_queries * d_code + gy * d_code + (offset + tx)
        ];
        cache_b[ty][tx] = table[
            gz * n_codewords * d_code + gx * d_code + (offset + ty)
        ];
        __syncthreads();

        // product
        for (index_t i = 0; i < BLOCK_SIZE; i += 1) {
            scalar_t v = (
                cache_a[ty][i] - cache_b[i][tx]
            ) > 0 ? value : -value;
            atomicAdd(&grad_query[
                gz * n_queries * d_code + gy * d_code + (offset + i)
            ], v);
            atomicAdd(&grad_table[
                gz * n_codewords * d_code + gx * d_code + (offset + i)
            ], -v);
        }
        __syncthreads();
    }
}
// clang-format on

torch::Tensor cdist_forward_cuda(
    const torch::Tensor &query, const torch::Tensor &table
) {
    CHECK_DIM(query, 3);
    CHECK_DIM(table, 3);
    TORCH_CHECK(query.size(0) == table.size(0));
    TORCH_CHECK(query.size(-1) == table.size(-1));
    TORCH_CHECK(query.scalar_type() == table.scalar_type());

    // sizes
    index_t d_code = table.size(-1);
    index_t n_queries = query.size(1);
    index_t n_codewords = table.size(1);
    index_t n_subspaces = table.size(0);
    TORCH_CHECK(d_code % TSZ == 0);
    TORCH_CHECK(n_queries % BSZ == 0);
    TORCH_CHECK(n_codewords % BSZ == 0);
    auto output = torch::zeros(
        {n_subspaces, n_queries, n_codewords}, query.options()
    );

    // dispatch
    dim3 threads(1, BSZ);
    dim3 blocks(1, n_queries / BSZ, n_subspaces);
    cdist_forward_kernel<float, float4><<<blocks, threads>>>(
        n_queries, n_codewords, d_code, query.data_ptr<float>(),
        table.data_ptr<float>(), output.data_ptr<float>()
    );
    TORCH_CHECK(hipGetLastError() == hipSuccess);

    //
    return output;
}

std::vector<torch::Tensor> cdist_backward_cuda(
    const torch::Tensor &query, const torch::Tensor &table,
    const torch::Tensor &grad_output
) {
    CHECK_DIM(query, 3);
    CHECK_DIM(table, 3);
    CHECK_DIM(grad_output, 3);
    TORCH_CHECK(query.size(0) == table.size(0));
    TORCH_CHECK(query.size(-1) == table.size(-1));
    TORCH_CHECK(query.size(0) == grad_output.size(0));
    TORCH_CHECK(query.size(1) == grad_output.size(1));
    TORCH_CHECK(table.size(1) == grad_output.size(-1));
    TORCH_CHECK(query.scalar_type() == table.scalar_type());
    TORCH_CHECK(query.scalar_type() == grad_output.scalar_type());

    // sizes
    index_t d_code = table.size(-1);
    index_t n_queries = query.size(1);
    index_t n_codewords = table.size(1);
    index_t n_subspaces = table.size(0);
    TORCH_CHECK(d_code % BLOCK_SIZE == 0);
    TORCH_CHECK(n_queries % BLOCK_SIZE == 0);
    TORCH_CHECK(n_codewords % BLOCK_SIZE == 0);
    auto grad_query = torch::zeros_like(query);
    auto grad_table = torch::zeros_like(table);

    // dispatch
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(n_codewords / BLOCK_SIZE, n_queries / BLOCK_SIZE, n_subspaces);
    AT_DISPATCH_FLOATING_TYPES(
        query.scalar_type(), "cdist_backward_kernel", ([&] {
            cdist_backward_kernel<scalar_t><<<blocks, threads>>>(
                n_queries, n_codewords, d_code, query.data_ptr<scalar_t>(),
                table.data_ptr<scalar_t>(), grad_output.data_ptr<scalar_t>(),
                grad_query.data_ptr<scalar_t>(), grad_table.data_ptr<scalar_t>()
            );
            TORCH_CHECK(hipGetLastError() == hipSuccess);
        })
    );

    //
    return {grad_query, grad_table};
}