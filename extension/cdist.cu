#include "hip/hip_runtime.h"
#include "common.h"

#define TSZ 4
#define BSZ 64

// clang-format off
template <typename scalar_t, typename vector_t>
__global__ void cdist_forward_kernel(
    index_t n_queries, index_t n_codewords, index_t d_code,
    const scalar_t *query, const scalar_t *table, scalar_t *output) {
    // index
    index_t ty = threadIdx.y;
    index_t gz = blockIdx.z * blockDim.z;
    index_t gy = blockIdx.y * blockDim.y + ty;

    // window
    for (index_t offset_x = 0; offset_x < n_codewords; offset_x += BSZ) {
        // reduce
        scalar_t reduced[BSZ] = {};
        for (index_t offset_k = 0; offset_k < d_code; offset_k += TSZ) {
            // cache
            __shared__ vector_t cache_query[BSZ];
            __shared__ vector_t cache_table[BSZ];
            cache_query[ty] = __ldg(
                (const vector_t *)&query[
                    gz * n_queries * d_code + gy * d_code + offset_k
                ]
            );
            cache_table[ty] = __ldg(
                (const vector_t *)&table[
                    gz * n_codewords * d_code + (offset_x + ty) * d_code + offset_k
                ]
            );
            __syncthreads();

            // product
            for (index_t tx = 0; tx < BSZ; tx += 1) {
                reduced[tx] += fabsf(cache_query[ty].x - cache_table[tx].x);
                reduced[tx] += fabsf(cache_query[ty].y - cache_table[tx].y);
                reduced[tx] += fabsf(cache_query[ty].z - cache_table[tx].z);
                reduced[tx] += fabsf(cache_query[ty].w - cache_table[tx].w);
            }
            __syncthreads();
        }

        // store
        index_t offset_z = gz * n_queries * n_codewords;
        for (index_t tx = 0; tx < BSZ; tx += TSZ) {
            index_t gx = offset_x + tx;
            __stcs(
                (vector_t *)&output[offset_z + gy * n_codewords + gx],
                *(const vector_t *)&reduced[tx]
            );
        }
    }
}

template <typename scalar_t, typename vector_t>
__global__ void cdist_backward_query_kernel(
    index_t n_queries, index_t n_codewords, index_t d_code,
    const scalar_t *query, const scalar_t *table, const scalar_t *grad_output,
    scalar_t *grad_query) {
    // index
    index_t ty = threadIdx.y;
    index_t gz = blockIdx.z * blockDim.z;
    index_t gy = blockIdx.y * blockDim.y + ty;

    // window
    for (index_t offset_k = 0; offset_k < d_code; offset_k += TSZ) {
        //cache
        __shared__ vector_t cache_query[BSZ];
        cache_query[ty] = __ldg(
            (const vector_t *)&query[
                gz * n_queries * d_code + gy * d_code + offset_k
            ]
        );

        // reduce
        vector_t reduced = {};
        for (index_t offset_x = 0; offset_x < n_codewords; offset_x += BSZ) {
            // cache
            __shared__ vector_t cache_table[BSZ];
            __shared__ scalar_t cache_grad_output[BSZ][BSZ];
            cache_table[ty] = __ldg(
                (const vector_t *)&table[
                    gz * n_codewords * d_code + (offset_x + ty) * d_code + offset_k
                ]
            );
            for (index_t tx = 0; tx < BSZ; tx += TSZ) {
                *(vector_t *)&cache_grad_output[ty][tx] = __ldg(
                    (const vector_t *)&grad_output[
                        gz * n_queries * n_codewords + gy * n_codewords + (offset_x + tx)
                    ]
                );
            }
            __syncthreads();

            // product
            for (index_t tx = 0; tx < BSZ; tx += 1) {
                scalar_t grad_v = cache_grad_output[ty][tx];
                vector_t grad_abs = {
                    (cache_query[ty].x - cache_table[tx].x) > 0 ? grad_v : -grad_v,
                    (cache_query[ty].y - cache_table[tx].y) > 0 ? grad_v : -grad_v,
                    (cache_query[ty].z - cache_table[tx].z) > 0 ? grad_v : -grad_v,
                    (cache_query[ty].w - cache_table[tx].w) > 0 ? grad_v : -grad_v,
                };
                reduced.x += grad_abs.x;
                reduced.y += grad_abs.y;
                reduced.z += grad_abs.z;
                reduced.w += grad_abs.w;
            }
            __syncthreads();
        }

        // store
        __stcs(
            (vector_t *)&grad_query[
                gz * n_queries * d_code + gy * d_code + offset_k
            ], reduced
        );
    }
}

template <typename scalar_t, typename vector_t>
__global__ void cdist_backward_table_kernel(
    index_t n_queries, index_t n_codewords, index_t d_code,
    const scalar_t *query, const scalar_t *table, const scalar_t *grad_output,
    scalar_t *grad_table) {
    // index
    index_t ty = threadIdx.y;
    index_t tx = threadIdx.x;
    index_t gz = blockIdx.z * blockDim.z;
    index_t gy = blockIdx.y * blockDim.y + ty;  // k
    index_t gx = blockIdx.x * blockDim.x + tx;  // n

    // cache
    __shared__ scalar_t cache_table[TSZ][BSZ];
    cache_table[ty][tx] = table[
        gz * n_codewords * d_code + gx * d_code + gy
    ];

    // window
    scalar_t reduced = 0.0;
    for (index_t offset_y = 0; offset_y < n_queries; offset_y += BSZ) {
        // cache
        __shared__ scalar_t cache_query[BSZ][TSZ];
        __shared__ scalar_t cache_grad_output[BSZ][BSZ];
        cache_query[tx][ty] = query[
            gz * n_queries * d_code + (offset_y + tx) * d_code + gy
        ];
        for (index_t i = 0; i < BSZ / TSZ; i += 1) {
            cache_grad_output[i * TSZ + ty][tx] = grad_output[
                gz * n_queries * n_codewords + (offset_y + i * TSZ + ty) * n_codewords + gx
            ];
        }
        __syncthreads();

        // product
        for (index_t i = 0; i < BSZ; i += 1) {
            scalar_t grad_v = cache_grad_output[i][tx];
            scalar_t grad_abs = (
                cache_query[i][ty] - cache_table[ty][tx]
            ) > 0 ? grad_v : -grad_v;
            reduced -= grad_abs;
        }
        __syncthreads();
    }

    // store
    grad_table[
        gz * n_codewords * d_code + gx * d_code + gy
    ] = reduced;
}
// clang-format on

torch::Tensor cdist_forward_cuda(
    const torch::Tensor &query, const torch::Tensor &table
) {
    CHECK_DIM(query, 3);
    CHECK_DIM(table, 3);
    CHECK_TYPE(query, torch::kFloat32);
    TORCH_CHECK(query.size(0) == table.size(0));
    TORCH_CHECK(query.size(-1) == table.size(-1));
    TORCH_CHECK(query.scalar_type() == table.scalar_type());

    // sizes
    index_t d_code = table.size(-1);
    index_t n_queries = query.size(1);
    index_t n_codewords = table.size(1);
    index_t n_subspaces = table.size(0);
    TORCH_CHECK(d_code % TSZ == 0);
    TORCH_CHECK(n_queries % BSZ == 0);
    TORCH_CHECK(n_codewords % BSZ == 0);
    auto output = torch::zeros(
        {n_subspaces, n_queries, n_codewords}, query.options()
    );

    // dispatch
    dim3 threads(1, BSZ);
    dim3 blocks(1, n_queries / BSZ, n_subspaces);
    cdist_forward_kernel<float, float4><<<blocks, threads>>>(
        n_queries, n_codewords, d_code, query.data_ptr<float>(),
        table.data_ptr<float>(), output.data_ptr<float>()
    );
    TORCH_CHECK(hipGetLastError() == hipSuccess);

    //
    return output;
}

std::vector<torch::Tensor> cdist_backward_cuda(
    const torch::Tensor &query, const torch::Tensor &table,
    const torch::Tensor &grad_output
) {
    CHECK_DIM(query, 3);
    CHECK_DIM(table, 3);
    CHECK_DIM(grad_output, 3);
    CHECK_TYPE(query, torch::kFloat32);
    TORCH_CHECK(query.size(0) == table.size(0));
    TORCH_CHECK(query.size(-1) == table.size(-1));
    TORCH_CHECK(query.size(0) == grad_output.size(0));
    TORCH_CHECK(query.size(1) == grad_output.size(1));
    TORCH_CHECK(table.size(1) == grad_output.size(-1));
    TORCH_CHECK(query.scalar_type() == table.scalar_type());
    TORCH_CHECK(query.scalar_type() == grad_output.scalar_type());

    // sizes
    index_t d_code = table.size(-1);
    index_t n_queries = query.size(1);
    index_t n_codewords = table.size(1);
    index_t n_subspaces = table.size(0);
    TORCH_CHECK(d_code % TSZ == 0);
    TORCH_CHECK(n_queries % BSZ == 0);
    TORCH_CHECK(n_codewords % BSZ == 0);
    auto grad_query = torch::zeros_like(query);
    auto grad_table = torch::zeros_like(table);

    // dispatch query
    {
        dim3 threads(1, BSZ);
        dim3 blocks(1, n_queries / BSZ, n_subspaces);
        cdist_backward_query_kernel<float, float4><<<blocks, threads>>>(
            n_queries, n_codewords, d_code, query.data_ptr<float>(),
            table.data_ptr<float>(), grad_output.data_ptr<float>(),
            grad_query.data_ptr<float>()
        );
        TORCH_CHECK(hipGetLastError() == hipSuccess);
    }

    // dispatch table
    {
        dim3 threads(BSZ, TSZ);
        dim3 blocks(n_codewords / BSZ, d_code / TSZ, n_subspaces);
        cdist_backward_table_kernel<float, float4><<<blocks, threads>>>(
            n_queries, n_codewords, d_code, query.data_ptr<float>(),
            table.data_ptr<float>(), grad_output.data_ptr<float>(),
            grad_table.data_ptr<float>()
        );
        TORCH_CHECK(hipGetLastError() == hipSuccess);
    }

    //
    return {grad_query, grad_table};
}