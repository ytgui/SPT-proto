#include "hip/hip_runtime.h"
#include "common.h"

#define BSZ 16

// clang-format off
template <typename scalar_t>
__global__ void softmax_forward_kernel(
    index_t seq_length, index_t nonzeros, const index_t *indptr,
    const index_t *indices, const scalar_t *values, scalar_t *output) {
    // index
    index_t ty = threadIdx.y;
    index_t gy = blockIdx.y * blockDim.y + ty;
    index_t gz = blockIdx.z * blockDim.z;

    // cumulate
    scalar_t cumulated = 0.0;
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += 1) {
        cumulated += expf(values[gz * nonzeros + i]);
    }
    cumulated = fmax(1e-9, cumulated);

    // softmax
    scalar_t scale = 1.0 / cumulated;
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += 1) {
        output[gz * nonzeros + i] = scale * expf(values[gz * nonzeros + i]);
    }
}

template <typename scalar_t, typename vector_t>
__global__ void softmax_backward_kernel(
    index_t seq_length, index_t nonzeros, const index_t *indptr,
    const index_t *indices, const scalar_t *values, const scalar_t *output,
    const scalar_t *grad_output, scalar_t *grad_values) {
    // index
    index_t ty = threadIdx.y;
    index_t gy = blockIdx.y * blockDim.y + ty;
    index_t gz = blockIdx.z * blockDim.z;

    // cumulate
    scalar_t cumulated = 0.0;
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += 1) {
        cumulated += output[gz * nonzeros + i] * grad_output[gz * nonzeros + i];
    }
    cumulated = fmax(1e-9, cumulated);

    // gradient
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += 1) {
        grad_values[gz * nonzeros + i] = output[gz * nonzeros + i] * (
            grad_output[gz * nonzeros + i] - cumulated
        );
    }
}
// clang-format on

torch::Tensor softmax_forward_cuda(
    const torch::Tensor &indptr, const torch::Tensor &indices,
    const torch::Tensor &values
) {
    CHECK_DIM(indptr, 1);
    CHECK_DIM(indices, 2);
    CHECK_DIM(values, 2);
    CHECK_TYPE(indptr, torch::kInt32);
    CHECK_TYPE(indices, torch::kInt32);
    TORCH_CHECK(indices.sizes() == values.sizes());

    // sizes
    index_t nonzeros = indices.size(-1);
    index_t batch_size = indices.size(0);
    index_t seq_length = indptr.size(-1) - 1;
    TORCH_CHECK((seq_length % BSZ) == 0);
    auto output = torch::empty_like(values);

    // dispatch
    dim3 threads(1, BSZ);
    dim3 blocks(1, seq_length / BSZ, batch_size);
    softmax_forward_kernel<float><<<blocks, threads>>>(
        seq_length, nonzeros, indptr.data_ptr<index_t>(),
        indices.data_ptr<index_t>(), values.data_ptr<float>(),
        output.data_ptr<float>()
    );
    TORCH_CHECK(hipGetLastError() == hipSuccess);

    //
    return output;
}

torch::Tensor softmax_backward_cuda(
    const torch::Tensor &indptr, const torch::Tensor &indices,
    const torch::Tensor &values, const torch::Tensor &output,
    const torch::Tensor &grad_output
) {
    CHECK_DIM(indptr, 1);
    CHECK_DIM(indices, 2);
    CHECK_DIM(values, 2);
    CHECK_DIM(output, 2);
    CHECK_DIM(grad_output, 2);
    CHECK_TYPE(indptr, torch::kInt32);
    CHECK_TYPE(indices, torch::kInt32);
    TORCH_CHECK(grad_output.sizes() == output.sizes());
    TORCH_CHECK(indices.sizes() == values.sizes());
    TORCH_CHECK(values.sizes() == output.sizes());

    // sizes
    index_t nonzeros = indices.size(-1);
    index_t batch_size = indices.size(0);
    index_t seq_length = indptr.size(-1) - 1;
    TORCH_CHECK((seq_length % BSZ) == 0);
    auto grad_values = torch::empty_like(values);

    // dispatch
    dim3 threads(1, BSZ);
    dim3 blocks(1, seq_length / BSZ, batch_size);
    softmax_backward_kernel<float, float4><<<blocks, threads>>>(
        seq_length, nonzeros, indptr.data_ptr<index_t>(),
        indices.data_ptr<index_t>(), values.data_ptr<float>(),
        output.data_ptr<float>(), grad_output.data_ptr<float>(),
        grad_values.data_ptr<float>()
    );
    TORCH_CHECK(hipGetLastError() == hipSuccess);

    //
    return grad_values;
}
