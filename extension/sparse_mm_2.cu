#include "hip/hip_runtime.h"
#include "common.h"

#define BK 16
#define BM 64
#define BN BM

template <typename scalar_t>
__global__ void sddmm_forward_cuda_kernel(
    index_t seq_length, index_t d_head, const index_t *indptr,
    const index_t *indices, const scalar_t *lhs, const scalar_t *rhs,
    scalar_t *output
) {
    // index
    index_t ty = threadIdx.y;
    index_t gy = blockIdx.y * blockDim.y + ty;

    // cache
    scalar_t cache_lhs[BK];
    __shared__ scalar_t cache_rhs[BN][BK];

    // k-loop
    for (index_t offset_k = 0; offset_k < d_head; offset_k += BK) {
        // sparse
        index_t cursor = indptr[gy];
        index_t cursor_limit = indptr[gy + 1];

        // n-loop
        for (index_t offset_n = 0; offset_n < seq_length; offset_n += BN) {
            // load
            for (index_t k = 0; k < BK; k += 1) {
                cache_lhs[k] = lhs[
                    gy * d_head + (offset_k + k)
                ];
                cache_rhs[ty][k] = rhs[
                    (offset_n + ty) * d_head + (offset_k + k)
                ];
            }
            __syncthreads();

            // contract
            while (cursor < cursor_limit) {
                index_t col = indices[cursor];
                if (col >= (offset_n + BN)) {
                    break;
                }

                // product
                scalar_t reduced = 0.0;
                for (index_t k = 0; k < BK; k += 1) {
                    reduced += cache_lhs[k] * cache_rhs[col % BN][k];
                }

                // store
                output[cursor] += reduced;
                cursor += 1;
            }
            __syncthreads();
        }
    }
}

torch::Tensor sddmm_forward_cuda(
    const torch::Tensor &indptr, const torch::Tensor &indices,
    const torch::Tensor &query, const torch::Tensor &key
) {
    CHECK_DIM(key, 2);
    CHECK_DIM(query, 2);
    CHECK_DIM(indptr, 1);
    CHECK_DIM(indices, 1);
    CHECK_TYPE(indptr, torch::kInt64);
    CHECK_TYPE(indices, torch::kInt64);
    TORCH_CHECK(query.sizes() == key.sizes());
    TORCH_CHECK(query.scalar_type() == key.scalar_type());

    // sizes
    index_t d_head = query.size(-1);
    TORCH_CHECK((d_head % BK) == 0);
    index_t seq_length = query.size(0);
    TORCH_CHECK((seq_length % BM) == 0);
    TORCH_CHECK(indptr.size(0) == seq_length + 1);
    auto output = torch::zeros_like(indices, query.options());

    // dispatch
    dim3 threads(1, BM);
    dim3 blocks(1, seq_length / BM);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        query.scalar_type(), "sddmm_forward_cuda_kernel", ([&] {
            sddmm_forward_cuda_kernel<scalar_t><<<blocks, threads>>>(
                seq_length, d_head, indptr.data_ptr<index_t>(),
                indices.data_ptr<index_t>(), query.data_ptr<scalar_t>(),
                key.data_ptr<scalar_t>(), output.data_ptr<scalar_t>()
            );
            TORCH_CHECK(hipGetLastError() == hipSuccess);
        })
    );

    //
    return output;
}