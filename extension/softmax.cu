#include "hip/hip_runtime.h"
#include "common.h"

#define TSZ 4
#define BSZ 16

// clang-format off
template <typename scalar_t, typename vector_t>
__global__ void softmax_forward_kernel(
    index_t seq_length, const index_t *indptr, const index_t *indices,
    const scalar_t *values, scalar_t *output) {
    // index
    index_t ty = threadIdx.y;
    index_t gy = blockIdx.y * blockDim.y + ty;

    // cumulate
    scalar_t cumulated = 0.0;
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += TSZ) {
        vector_t cache = __ldg(
            (const vector_t *)&values[i]
        );
        cumulated += expf(cache.x) + expf(cache.y);
        cumulated += expf(cache.z) + expf(cache.w);
    }

    // softmax
    scalar_t scale = 1.0 / cumulated;
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += TSZ) {
        vector_t cache = __ldg(
            (const vector_t *)&values[i]
        );
        cache.x = scale * expf(cache.x);
        cache.y = scale * expf(cache.y);
        cache.z = scale * expf(cache.z);
        cache.w = scale * expf(cache.w);
        __stcs((vector_t *)&output[i], cache);
    }
}

template <typename scalar_t, typename vector_t>
__global__ void softmax_backward_kernel(
    index_t seq_length, const index_t *indptr, const index_t *indices,
    const scalar_t *values, const scalar_t *output, const scalar_t *grad_output,
    scalar_t *grad_values) {
    // index
    index_t ty = threadIdx.y;
    index_t gy = blockIdx.y * blockDim.y + ty;

    // cumulate
    scalar_t cumulated = 0.0;
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += TSZ) {
        vector_t cache_output = __ldg(
            (const vector_t *)&output[i]
        );
        vector_t cache_grad_output = __ldg(
            (const vector_t *)&grad_output[i]
        );
        cumulated += cache_output.x * cache_grad_output.x;
        cumulated += cache_output.y * cache_grad_output.y;
        cumulated += cache_output.z * cache_grad_output.z;
        cumulated += cache_output.w * cache_grad_output.w;
    }

    // gradient
    for (index_t i = indptr[gy]; i < indptr[gy + 1]; i += TSZ) {
        vector_t cache_output = __ldg(
            (const vector_t *)&output[i]
        );
        vector_t cache_grad_output = __ldg(
            (const vector_t *)&grad_output[i]
        );
        cache_output.x = cache_output.x * (
            cache_grad_output.x - cumulated
        );
        cache_output.y = cache_output.y * (
            cache_grad_output.y - cumulated
        );
        cache_output.z = cache_output.z * (
            cache_grad_output.z - cumulated
        );
        cache_output.w = cache_output.w * (
            cache_grad_output.w - cumulated
        );
        __stcs((vector_t *)&grad_values[i], cache_output);
    }
}
// clang-format on

torch::Tensor softmax_forward_cuda(
    const torch::Tensor &indptr, const torch::Tensor &indices,
    const torch::Tensor &values
) {
    CHECK_DIM(indptr, 1);
    CHECK_DIM(indices, 1);
    CHECK_DIM(values, 1);
    CHECK_TYPE(indptr, torch::kInt32);
    CHECK_TYPE(indices, torch::kInt32);
    TORCH_CHECK(indices.sizes() == values.sizes());

    // sizes
    index_t seq_length = indptr.size(0) - 1;
    TORCH_CHECK((seq_length % BSZ) == 0);
    auto output = torch::empty_like(values);

    // dispatch
    dim3 threads(1, BSZ);
    dim3 blocks(1, seq_length / BSZ);
    softmax_forward_kernel<float, float4><<<blocks, threads>>>(
        seq_length, indptr.data_ptr<index_t>(), indices.data_ptr<index_t>(),
        values.data_ptr<float>(), output.data_ptr<float>()
    );
    TORCH_CHECK(hipGetLastError() == hipSuccess);

    //
    return output;
}

torch::Tensor softmax_backward_cuda(
    const torch::Tensor &indptr, const torch::Tensor &indices,
    const torch::Tensor &values, const torch::Tensor &output,
    const torch::Tensor &grad_output
) {
    CHECK_DIM(indptr, 1);
    CHECK_DIM(indices, 1);
    CHECK_DIM(values, 1);
    CHECK_DIM(output, 1);
    CHECK_DIM(grad_output, 1);
    CHECK_TYPE(indptr, torch::kInt32);
    CHECK_TYPE(indices, torch::kInt32);
    TORCH_CHECK(indices.sizes() == values.sizes());
    TORCH_CHECK(values.sizes() == output.sizes());

    // sizes
    index_t seq_length = indptr.size(0) - 1;
    TORCH_CHECK((seq_length % BSZ) == 0);
    auto grad_values = torch::empty_like(values);

    // dispatch
    dim3 threads(1, BSZ);
    dim3 blocks(1, seq_length / BSZ);
    softmax_backward_kernel<float, float4><<<blocks, threads>>>(
        seq_length, indptr.data_ptr<index_t>(), indices.data_ptr<index_t>(),
        values.data_ptr<float>(), output.data_ptr<float>(),
        grad_output.data_ptr<float>(), grad_values.data_ptr<float>()
    );
    TORCH_CHECK(hipGetLastError() == hipSuccess);

    //
    return grad_values;
}
